#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <memory>
#include <cassert>
#include <cstdlib>
#include <ctime>
#include <algorithm>
#include <numeric>
#include <set>

//const int TILE_DIM = 32;
//const int BLOCK_ROWS = 8;

template <typename Node, typename Weight>
struct Edge {
  Node   s;
  Node   t;
  Weight w;
  Edge(Node s, Node t, Weight w) : s(s), t(t), w(w) {}
};

template <typename Node, typename Weight>
struct Csr {
  std::vector<unsigned> adj_ptr;
  std::vector<Node> adj_node;
  std::vector<Weight> adj_wgt;
  std::vector<Weight> deg;
};

template <typename Node, typename Weight>
struct Graph {
  long N; // number of node and edge
  long E;
  std::vector<Edge<Node, Weight>> edges;
  Csr<Node, Weight> csr_out;
  Csr<Node, Weight> csr_in;
  std::vector<Weight> deg;
  std::vector<Node> blocks;
  Graph() : N(0), E(0) {}
};

template <typename Node, typename Weight>
struct Block {
  Csr<Node, Weight> csr_out;
  Csr<Node, Weight> csr_in;
  std::vector<Weight> deg;
};

template <typename Node, typename Weight>
bool compare_s(const Edge<Node, Weight>& e1, const Edge<Node, Weight>& e2) {
  return e1.s < e2.s;
}

template <typename Node, typename Weight>
bool compare_t(const Edge<Node, Weight>& e1, const Edge<Node, Weight>& e2) {
  return e1.t < e2.t;
}

template <typename Node, typename Weight>
Graph<Node, Weight> load_graph_from_tsv(const std::string& FileName) {
  std::ifstream file(FileName + ".tsv");
  if (!file.is_open()) {
    std::cerr << "Unable to open file!\n";
    std::exit(EXIT_FAILURE);
  }
  std::string line; // format: node i \t node j \t  w_ij
  std::vector<std::string> v_line;

  Graph<Node, Weight> g;
  Node s, t;
  Weight w;
  unsigned start, tab_pos;
  while (std::getline(file, line)) {
    start = 0;
    tab_pos = line.find('\t');
    s = static_cast<Node>(std::stoi(line.substr(start, tab_pos - start)));
    start = tab_pos + 1;
    tab_pos = line.find('\t', start);
    t = static_cast<Node>(std::stoi(line.substr(start, tab_pos - start)));
    start = tab_pos + 1;
    tab_pos = line.find('\t', start);
    w = static_cast<Weight>(std::stof(line.substr(start, tab_pos - start)));
    g.edges.emplace_back(s-1, t-1, w);
    if (s > g.N) g.N = s; 
  }
  file.close();

  g.E = g.edges.size();
  g.deg.resize(g.N);
  
  std::sort(g.edges.begin(), g.edges.end(), compare_s<Node, Weight>);
  g.csr_out.adj_ptr.emplace_back(0);
  g.csr_out.deg.resize(g.N);
  s = 0;
  for (unsigned i = 0; i < g.E; i++) {
    if (g.edges[i].s != s) {
      s++;
      g.csr_out.adj_ptr.emplace_back(i);
    }
    g.csr_out.adj_node.emplace_back(g.edges[i].t);
    g.csr_out.adj_wgt.emplace_back(g.edges[i].w);
    g.csr_out.deg[g.edges[i].s] += g.edges[i].w;
    g.deg[g.edges[i].s] += g.edges[i].w;
  }

  std::sort(g.edges.begin(), g.edges.end(), compare_t<Node, Weight>);
  g.csr_in.adj_ptr.emplace_back(0);
  g.csr_in.deg.resize(g.N);
  t = 0;
  for (unsigned i = 0; i < g.E; i++) {
    if (g.edges[i].t != t) {
      t++;
      g.csr_in.adj_ptr.emplace_back(i);
    }
    g.csr_in.adj_node.emplace_back(g.edges[i].s);
    g.csr_in.adj_wgt.emplace_back(g.edges[i].w);
    g.csr_in.deg[g.edges[i].t] += g.edges[i].w;
    g.deg[g.edges[i].t] += g.edges[i].w;
  }
  return g;
}


// ----------------------CUDA kernel ---------------------//

template <typename Node, typename Weight>
__global__ void random_block_generator(Node* gpu_random_blocks, unsigned B) {

  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < B) {
    hiprandState state;
    hiprand_init(clock64(), idx, 0, &state);
    gpu_random_blocks[idx] = hiprand(&state) % B;
  }

}

template <typename Node, typename Weight>
__global__ void random_block_generator_nodal(Node* gpu_random_blocks, unsigned B, unsigned N) {

  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    hiprandState state;
    hiprand_init(clock64(), idx, 0, &state);
    gpu_random_blocks[idx] = hiprand(&state) % B;
  }

}


template <typename Node, typename Weight>
__global__ void uniform_number_generator(float* gpu_uniform_x, unsigned B) {

  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < B) {
    hiprandState state;
    hiprand_init(clock64(), idx, 0, &state);
    gpu_uniform_x[idx] = hiprand_uniform(&state);
  }

}

template <typename Node, typename Weight>
__global__ void calculate_acceptance_prob(float* gpu_acceptance_prob, Weight*
                                          deg, unsigned B) {

  unsigned idx =  blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < B) {
    gpu_acceptance_prob[idx] = (float)B/(deg[idx]+B);
  }

}

template <typename Node, typename Weight>
__global__ void sample_neighbors(Node* gpu_sampling_neighbors, 
                                 unsigned* csr_out_adj_ptr,
  				 Node* csr_out_adj_node,
                                 Weight* csr_out_adj_wgt,
                                 unsigned* csr_in_adj_ptr,
                                 Node* csr_in_adj_node,
                                 Weight* csr_in_adj_wgt,
                                 Weight* deg, unsigned B) {

  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < B) {
    
    hiprandState state;
    hiprand_init(clock64(), idx, 0, &state);
    float random = hiprand_uniform(&state);

    unsigned out_s, out_e;
    unsigned in_s, in_e;
    out_s = csr_out_adj_ptr[idx];
    in_s = csr_in_adj_ptr[idx];
    if (idx + 1 < B) {
      out_e = csr_out_adj_ptr[idx+1];
      in_e = csr_in_adj_ptr[idx+1];
    }
    else {
      out_e = B;
      in_e = B; 
    }

    float prob_sum = 0.0f;
    unsigned neighbor;
    unsigned find = 0;
    for (unsigned i = out_s; i < out_e; i++) {
      prob_sum += (float)csr_out_adj_wgt[i]/deg[idx];
      if (random > prob_sum) {
        neighbor = csr_out_adj_node[i];
        find = 1;
        break;
      }
    }
    if (find == 0) {
      for (unsigned i = in_s; i < in_e; i++) {
        prob_sum += (float)csr_in_adj_wgt[i]/deg[idx];
        if (random > prob_sum) {
          neighbor = csr_in_adj_node[i];
          break;
        }
      }
    }

    gpu_sampling_neighbors[idx] = neighbor;
     
  }

}


template <typename Node, typename Weight>
__global__ void sample_neighbors_nodal(Node* gpu_sampling_neighbors,
                                       unsigned* csr_out_adj_ptr,
                                       Node* csr_out_adj_node,
                                       Weight* csr_out_adj_wgt,
                                       unsigned* csr_in_adj_ptr,
                                       Node* csr_in_adj_node,
                                       Weight* csr_in_adj_wgt,
                                       Weight* deg, 
				       Node* blocks, unsigned N) {

  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    
    hiprandState state;
    hiprand_init(clock64(), idx, 0, &state);
    float random = hiprand_uniform(&state);

    unsigned out_s, out_e;
    unsigned in_s, in_e;

    out_s = csr_out_adj_ptr[idx];
    in_s = csr_in_adj_ptr[idx];
    if (idx + 1 < N) {
      out_e = csr_out_adj_ptr[idx+1];
      in_e = csr_in_adj_ptr[idx+1];
    }
    else {
      out_e = N;
      in_e = N;
    }

    float prob_sum = 0.0f;
    unsigned neighbor;
    unsigned find = 0;
    for (unsigned i = out_s; i < out_e; i++) {
      prob_sum += (float)csr_out_adj_wgt[i]/deg[idx];
      if (random > prob_sum) {
        neighbor = blocks[csr_out_adj_node[i]];
        find = 1;
        break;
      }
    }
    if (find == 0) {
      for (unsigned i = in_s; i < in_e; i++) {
        prob_sum += (float)csr_in_adj_wgt[i]/deg[idx];
        if (random > prob_sum) {
          neighbor = blocks[csr_in_adj_node[i]];
          break;
        }
      }
    }

    gpu_sampling_neighbors[idx] = neighbor;
  
  }

}



template <typename Node, typename Weight>
__global__ void calculate_dS_out(float* dS_out,
	       			 unsigned* csr_out_adj_ptr,
  				 Node* csr_out_adj_node,
  				 Weight* csr_out_adj_wgt,
  				 Weight* csr_out_deg,
  				 Weight* csr_in_deg,
                                 unsigned B) {
  
  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < B) {
    unsigned out_s = csr_out_adj_ptr[idx];
    unsigned out_e;
    if (idx + 1 < B) {
      out_e = csr_out_adj_ptr[idx+1];
    }
    else {
      out_e = B;
    }
    float dS = 0;
    for (unsigned i = out_s; i < out_e; i++) {
      dS += (float)csr_out_adj_wgt[i] * std::log((float)csr_out_adj_wgt[i]
          / (csr_out_deg[idx] * csr_in_deg[csr_out_adj_node[i]]));
    }
    dS_out[idx] = dS;
  }

}

template <typename Node, typename Weight>
__global__ void calculate_dS_in(float* dS_in, 
                                unsigned* csr_in_adj_ptr,
                                Node* csr_in_adj_node,
                                Weight* csr_in_adj_wgt,
                                Weight* csr_in_deg,
                                Weight* csr_out_deg,
				unsigned B) {
    
  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < B) {
    unsigned in_s = csr_in_adj_ptr[idx];
    unsigned in_e;
    if (idx + 1 < B) {
      in_e = csr_in_adj_ptr[idx+1];
    }
    else {
      in_e = B;
    }
    float dS = 0;
    for (unsigned i = in_s; i < in_e; i++) {
      dS += (float)csr_in_adj_wgt[i] * std::log((float)csr_in_adj_wgt[i]
          / (csr_out_deg[csr_in_adj_node[i]] * csr_in_deg[idx]));
    }
    dS_in[idx] = dS;
  }

}


// Potential Bottleneck?
template <typename Node, typename Weight>
__global__ void calculate_dS_new_out(float* dS_new_out, Node* s,
                                     unsigned* csr_out_adj_ptr,
                                     Node* csr_out_adj_node,
                                     Weight* csr_out_adj_wgt,
                                     Weight* csr_out_deg,
                                     Weight* csr_in_deg, 
				     unsigned B) {

  unsigned r = blockIdx.x * blockDim.x + threadIdx.x;

  if (r < B) {
    unsigned s_ = s[r]; 
    unsigned out_r_s, out_r_e;
    unsigned out_s_s, out_s_e;
    out_r_s = csr_out_adj_ptr[r];
    out_s_s = csr_out_adj_ptr[s_];
    if (r + 1 < B) {
      out_r_e = csr_out_adj_ptr[r+1];
    }   
    else {
      out_r_e = B;
    }    
    if (s_ + 1 < B) {
      out_s_e = csr_out_adj_ptr[s_+1];
    }
    else {
      out_s_e = B;
    }
    unsigned i = out_r_s, j = out_s_s;
    float dS = 0;
    Weight w;
    Weight dout = csr_out_deg[r] + csr_out_deg[s_];
    Node n;
    // ascending order
    while (i < out_r_e && j < out_s_e) {
      if (csr_out_adj_node[i] < csr_out_adj_node[j]) {
        w = csr_out_adj_wgt[i];
        n = csr_out_adj_node[i];
        i++;
      }
      else if (csr_out_adj_node[i] > csr_out_adj_node[j]) {
        w = csr_out_adj_wgt[j];
        n = csr_out_adj_node[j];
        j++;
      }
      else {
        w = csr_out_adj_wgt[i] + csr_out_adj_wgt[j];
        n = csr_out_adj_node[i];
        i++;
        j++;
      }
      dS -= w * std::log((float)w/(dout*csr_in_deg[n]));
    }
    for (; i < out_r_e; i++) {
      w = csr_out_adj_wgt[i];
      n = csr_out_adj_node[i];
      dS -= w * std::log((float)w/(dout*csr_in_deg[n]));
    }
    for (; j < out_s_e; j++) {
      w = csr_out_adj_wgt[j];
      n = csr_out_adj_node[j];
      dS -= w * std::log((float)w/(dout*csr_out_deg[n]));
    }
    dS_new_out[r] = dS;
  }

}


template <typename Node, typename Weight>
__global__ void calculate_dS_new_in(float* dS_new_in, Node* s,
                                    unsigned* csr_in_adj_ptr,
                                    Node* csr_in_adj_node,
                                    Weight* csr_in_adj_wgt,
                                    Weight* csr_in_deg,                    
				    Weight* csr_out_deg,
				    unsigned B) {

  unsigned r = blockIdx.x * blockDim.x + threadIdx.x;

  if (r < B) {
    unsigned s_ = s[r];
    unsigned in_r_s, in_r_e;
    unsigned in_s_s, in_s_e;
    in_r_s = csr_in_adj_ptr[r];
    in_s_s = csr_in_adj_ptr[s_];
    if (r + 1 < B) {
      in_r_e = csr_in_adj_ptr[r+1];
    } 
    else {
      in_r_e = B;
    } 
    if (s_ + 1 < B) {
      in_s_e = csr_in_adj_ptr[s_+1];
    }
    else {
      in_s_e = B;
    }
    unsigned i = in_r_s, j = in_s_s;
    float dS = 0;
    Weight w;
    Weight din = csr_in_deg[r] + csr_in_deg[s_];
    Node n;
    // ascending order
    while (i < in_r_e && j < in_s_e) {
      if (csr_in_adj_node[i] < csr_in_adj_node[j]) {
        w = csr_in_adj_wgt[i];
        n = csr_in_adj_node[i];
        i++;
      }
      else if (csr_in_adj_node[i] > csr_in_adj_node[j]) {
        w = csr_in_adj_wgt[j];
        n = csr_in_adj_node[j];
        j++;
      }
      else {
        w = csr_in_adj_wgt[i] + csr_in_adj_wgt[j];
        n = csr_in_adj_node[i];
        i++;
        j++;
      }
      dS -= w * std::log((float)w/(csr_out_deg[n]*din));
    }
    for (; i < in_r_e; i++) {
      w = csr_in_adj_wgt[i];
      n = csr_in_adj_node[i];
      dS -= w * std::log((float)w/(csr_out_deg[n]*din));
    }
    for (; j < in_s_e; j++) {
      w = csr_in_adj_wgt[j];
      n = csr_in_adj_node[j];
      dS -= w * std::log((float)w/(csr_out_deg[n]*din));
    }
    dS_new_in[r] = dS;
  }

}

template <typename Node, typename Weight>
__global__ void calculate_dS_overall(float* dS, float* dS_out, float* dS_in,
                                     float* dS_new_out, float* dS_new_in, 
				     unsigned* csr_out_adj_ptr,
                                     Node* csr_out_adj_node,
                                     Weight* csr_out_adj_wgt,
                                     Weight* csr_out_deg,
                                     Weight* csr_in_deg,
				     Node* s,
                                     Node* bestS,
				     unsigned B) {

  unsigned r = blockIdx.x * blockDim.x + threadIdx.x;

  if (r < B) {
    unsigned s_ = s[r];
    float dS_ = 0;
    dS_ += dS_out[r];
    dS_ += dS_out[s_];
    dS_ += dS_in[r];
    dS_ += dS_in[s_];
    dS_ -= dS_new_out[r];
    dS_ -= dS_new_in[r];
    
    unsigned out_r_s = csr_out_adj_ptr[r];
    unsigned out_r_e;
    unsigned out_s_s = csr_out_adj_ptr[s_];
    unsigned out_s_e;
    if (r + 1 < B) {
      out_r_e = csr_out_adj_ptr[r+1];
    } 
    else {
      out_r_e = B;
    }
    if (s_ + 1 < B) {
      out_s_e = csr_out_adj_ptr[s_+1];
    }
    else {
      out_s_e = B;
    }
    for (unsigned i = out_r_s; i < out_r_e; i++) {
      if (csr_out_adj_node[i] == r) {
        dS_ -= csr_out_adj_wgt[i]
          * std::log((float)csr_out_adj_wgt[i]/(csr_out_deg[r]*csr_in_deg[r]));
      }
      if (csr_out_adj_node[i] == s_) {
        dS_ -= csr_out_adj_wgt[i]
          * std::log((float)csr_out_adj_wgt[i]/(csr_out_deg[r]*csr_in_deg[s_]));
      }
    }
    for (unsigned i = out_s_s; i < out_s_e; i++) {
      if (csr_out_adj_node[i] == r) {
        dS_ -= csr_out_adj_wgt[i]
          * std::log((float)csr_out_adj_wgt[i]/(csr_out_deg[s_]*csr_in_deg[r]));
      }
      if (csr_out_adj_node[i] == s_) {
        dS_ -= csr_out_adj_wgt[i]
          * std::log((float)csr_out_adj_wgt[i]/(csr_out_deg[s_]*csr_in_deg[s_]));
      }
    }
    //dS[r] = dS_;
    if (dS_ < dS[r]) {
      bestS[r] = s_;
      dS[r] = dS_;
    }
  }
}


template <typename Node, typename Weight>
__global__ void calculate_dS_new_out_nodal(float* dS_new_out, Node* s,
                                           unsigned* g_csr_out_adj_ptr,
                                           Node* g_csr_out_adj_node,
                                           unsigned* b_csr_out_adj_ptr,
                                           Node* b_csr_out_adj_node,

					   Weight* b_csr_out_adj_wgt,
                                           Weight* b_csr_out_deg,
                                           
					   
					   
					   Weight* csr_in_deg,
                                           
					   
					   
					   Node* blocks,
					   unsigned N) {

  
  unsigned ni = blockIdx.x * blockDim.x + threadIdx.x;

  if (ni < N) {
   
    unsigned r = blocks[ni];

    unsigned node_e;
    if (ni + 1 < N) {
      node_e = g_csr_out_adj_ptr[ni+1];    	
    }
    else {
      node_e = N;
    }
    Node b;
    for (unsigned i = node_s; i < node_e; i++) {
      b = blocks[g_csr_out_adj_node[i]];
    
    }
  
  }
}


template <typename Node, typename Weight>
__global__ void propose(Node* gpu_random_blocks, Node* gpu_sampling_neighbors1,
                        Node* gpu_sampling_neighbors2, Node* s, 
                        float* gpu_uniform_x, float* gpu_acceptance_prob,
                        Weight* deg, unsigned B) {

  unsigned r = blockIdx.x * blockDim.x + threadIdx.x;

  if (r < B) {
    
    if (deg[r] > 0) {
      Node u = gpu_sampling_neighbors1[r];
      if (gpu_uniform_x[r] <= gpu_acceptance_prob[u]) {
        s[r] = gpu_random_blocks[r];
      }
      else {
        Node u = gpu_sampling_neighbors2[r];
        if (deg[u] == 0) {
          s[r] = gpu_random_blocks[r];  // Should be different?
        }
        else {
          s[r] = u;
        }
      }
    }
    else {
      s[r] = gpu_random_blocks[r];
    }
  }

}

template <typename Node, typename Weight>
__global__ void propose_n(Node* gpu_random_blocks, Node* gpu_sampling_neighbors,
                          Node* gpu_sampling_neighbors_nodal, Node* s,
                          float* gpu_uniform_x, float* gpu_acceptance_prob,
                          Weight* node_deg, unsigned N) {

  unsigned ni = blockIdx.x * blockDim.x + threadIdx.x;

  if (ni < N) {
    if (node_deg[ni] > 0) {
      Node u = gpu_sampling_neighbors_nodal[ni];
      if (gpu_uniform_x[ni] <= gpu_acceptance_prob[u]) {
        s[ni] = gpu_random_blocks[ni];
      }
      else {
        s[ni] = gpu_sampling_neighbors[u];
      }
    }
    else {
      s[ni] = gpu_random_blocks[ni];
    }  
  }

}


// ---------------------- Partition -------------------------//
template <typename Node, typename Weight>
void initialize_block_count(Graph<Node, Weight>& g, Block<Node, Weight>& b, unsigned B) {

  std::vector<std::vector<std::pair<Node, Weight>>> Mrow(B);
  std::vector<std::vector<std::pair<Node, Weight>>> Mcol(B);

  b.deg.clear();
  b.deg.resize(B);
  for (unsigned i = 0; i < g.N; i++) {
    Node k1 = g.blocks[i];
    unsigned end;
    if (k1 + 1 < B) {
      end = g.csr_out.adj_ptr[k1+1];
    }
    else {
      end = B;
    }
    for(unsigned j = g.csr_out.adj_ptr[k1]; j < end; j++) {
      Node k2 = g.blocks[g.csr_out.adj_node[j]];
      Mrow[k1].emplace_back(k2, g.csr_out.adj_wgt[j]);
      b.deg[k1] += g.csr_out.adj_wgt[j];
    }
  }

  b.csr_out.adj_ptr.clear();
  b.csr_out.deg.resize(g.N);
  unsigned ptr = 0;
  for (unsigned i = 0; i < g.N; i++) {
    b.csr_out.adj_ptr.emplace_back(ptr);
    for (const auto& [j, w] : Mrow[i]) {
      b.csr_out.adj_node.emplace_back(j);
      b.csr_out.adj_wgt.emplace_back(w);
      b.csr_out.deg[i] += w;
      ptr++;
    }
  }

  for (unsigned i = 0; i < g.N; i++) {
    Node k1 = g.blocks[i];
    unsigned end;
    if (k1 + 1 < B) {
      end = g.csr_in.adj_ptr[k1+1];
    }
    else {
      end = B;
    }
    for(unsigned j = g.csr_in.adj_ptr[k1]; j < end; j++) {
      Node k2 = g.blocks[g.csr_in.adj_node[j]];
      Mcol[k1].emplace_back(k2, g.csr_in.adj_wgt[j]);
      b.deg[k1] += g.csr_in.adj_wgt[j];
    }
  }

  b.csr_in.adj_ptr.clear();
  b.csr_in.deg.resize(g.N);
  ptr = 0;
  for (unsigned i = 0; i < g.N; i++) {
    b.csr_in.adj_ptr.emplace_back(ptr);
    for (const auto& [j, w] : Mcol[i]) {
      b.csr_in.adj_node.emplace_back(j);
      b.csr_in.adj_wgt.emplace_back(w);
      b.csr_in.deg[i] += w;
      ptr++;
    }
  }
  

}



template <typename Node, typename Weight>
void carry_out_best_merge(Graph<Node, Weight>& g, 
		          std::vector<float>& dS, 
			  std::vector<Node>& S, 
			  unsigned B, unsigned bToMerges) {

  std::vector<Node> bestMerges(B);
  std::vector<int> blockMap(B);
  std::vector<Node> remainBlocks;
  std::set<Node> seen;

  std::iota(bestMerges.begin(), bestMerges.end(), 0);
  std::iota(blockMap.begin(), blockMap.end(), 0);
  std::sort(bestMerges.begin(), bestMerges.end(), [&] (unsigned i, unsigned j) {
    return dS[i] < dS[j];
  });

  unsigned numMerges = 0;
  unsigned counter = 0;
  while (numMerges < bToMerges) {
    Node mergeFrom = bestMerges[counter];
    Node mergeTo = blockMap[S[mergeFrom]];
    counter++;
    if (mergeTo != mergeFrom) {
      for (unsigned i = 0; i < B; i++) {
      	if (blockMap[i] == mergeFrom) {
	  blockMap[i] = mergeTo;
	}
      }
      for (unsigned i = 0; i < B; i++) {
      	if (g.blocks[i] == mergeFrom) {
	  g.blocks[i] = mergeTo;
	}
      }
      numMerges++;
    }
  }

  for (const auto& b : g.blocks) {
    if (seen.find(b) == seen.end()) {
      seen.insert(b);
    }
  }

  remainBlocks.insert(remainBlocks.end(), seen.begin(), seen.end());
  std::sort(remainBlocks.begin(), remainBlocks.end());
  blockMap.clear();
  blockMap.resize(B, -1);
  for (unsigned i = 0; i < remainBlocks.size(); i++) {
    blockMap[remainBlocks[i]] = i;
  }
  for (auto& b : g.blocks) {
    b = blockMap[b];
  }

}


template <typename Node, typename Weight>
void propose_block_merge(Block<Node, Weight>& b, unsigned B, 
		         std::vector<Node>& S,
	                 std::vector<float>& dS,
			 unsigned numProposals) {
  
  unsigned block_size = 256;
  unsigned num_blocks = (B + block_size - 1) / block_size;
 
  // Create stream
  hipStream_t s1, s2, s3, s4, s5;
  hipStreamCreate(&s1);
  hipStreamCreate(&s2);
  hipStreamCreate(&s3);
  hipStreamCreate(&s4);
  hipStreamCreate(&s5);

  // device data
  unsigned* gpu_csr_out_adj_ptr;
  Node* gpu_csr_out_adj_node;
  Weight* gpu_csr_out_adj_wgt;
  Weight* gpu_csr_out_deg;

  unsigned* gpu_csr_in_adj_ptr;
  Node* gpu_csr_in_adj_node;
  Weight* gpu_csr_in_adj_wgt;
  Weight* gpu_csr_in_deg;

  Weight* gpu_deg;
  Node* gpu_random_blocks;
  Node* gpu_sampling_neighbors1;
  Node* gpu_sampling_neighbors2;
  float* gpu_uniform_x;
  float* gpu_acceptance_prob;
  Node* proposed_blocks;
  Node* best_proposed_blocks;
  float* gpu_dS_out;
  float* gpu_dS_in;
  float* gpu_dS_new_out;
  float* gpu_dS_new_in;
  float* gpu_dS;


  // Allocate GPU memory
  hipMallocAsync(&gpu_csr_out_adj_ptr, sizeof(unsigned)*b.csr_out.adj_ptr.size(), s1);  
  hipMallocAsync(&gpu_csr_out_adj_node, sizeof(Node)*b.csr_out.adj_node.size(), s1);
  hipMallocAsync(&gpu_csr_out_adj_wgt, sizeof(Weight)*b.csr_out.adj_wgt.size(), s1);
  hipMallocAsync(&gpu_csr_out_deg, sizeof(Weight)*b.csr_out.deg.size(), s1);
  hipMallocAsync(&gpu_csr_in_adj_ptr, sizeof(unsigned)*b.csr_in.adj_ptr.size(), s1);
  hipMallocAsync(&gpu_csr_in_adj_node, sizeof(Node)*b.csr_in.adj_node.size(), s1);
  hipMallocAsync(&gpu_csr_in_adj_wgt, sizeof(Weight)*b.csr_in.adj_wgt.size(), s1);
  hipMallocAsync(&gpu_csr_in_deg, sizeof(Weight)*b.csr_in.deg.size(), s1);
  hipMallocAsync(&gpu_deg, sizeof(Weight)*B, s1);
  hipMallocAsync(&gpu_random_blocks, sizeof(Node)*B, s1);
  hipMallocAsync(&gpu_sampling_neighbors1, sizeof(Node)*B, s1);
  hipMallocAsync(&gpu_sampling_neighbors2, sizeof(Node)*B, s1);
  hipMallocAsync(&gpu_uniform_x, sizeof(float)*B, s1);
  hipMallocAsync(&gpu_acceptance_prob, sizeof(float)*B, s1);
  hipMallocAsync(&proposed_blocks, sizeof(Node)*B, s1);
  hipMallocAsync(&best_proposed_blocks, sizeof(Node)*B, s1);
  hipMallocAsync(&gpu_dS_out, sizeof(float)*B, s1);
  hipMallocAsync(&gpu_dS_in, sizeof(float)*B, s1);
  hipMallocAsync(&gpu_dS_new_out, sizeof(float)*B, s1);
  hipMallocAsync(&gpu_dS_new_in, sizeof(float)*B, s1);
  hipMallocAsync(&gpu_dS, sizeof(float)*B, s1);


  // transfer data
  hipMemcpyAsync(gpu_csr_out_adj_ptr, b.csr_out.adj_ptr.data(), sizeof(unsigned)*b.csr_out.adj_ptr.size(), hipMemcpyDefault, s1);
  hipMemcpyAsync(gpu_csr_out_adj_node, b.csr_out.adj_node.data(), sizeof(Node)*b.csr_out.adj_node.size(), hipMemcpyDefault, s1);
  hipMemcpyAsync(gpu_csr_out_adj_wgt, b.csr_out.adj_wgt.data(), sizeof(Weight)*b.csr_out.adj_wgt.size(), hipMemcpyDefault, s1);
  hipMemcpyAsync(gpu_csr_out_deg, b.csr_out.deg.data(), sizeof(Weight)*b.csr_out.deg.size(), hipMemcpyDefault, s1);
  hipMemcpyAsync(gpu_csr_in_adj_ptr, b.csr_in.adj_ptr.data(), sizeof(unsigned)*b.csr_in.adj_ptr.size(), hipMemcpyDefault, s1);
  hipMemcpyAsync(gpu_csr_in_adj_node, b.csr_in.adj_node.data(), sizeof(Node)*b.csr_in.adj_node.size(), hipMemcpyDefault, s1);
  hipMemcpyAsync(gpu_csr_in_adj_wgt, b.csr_in.adj_wgt.data(), sizeof(Weight)*b.csr_in.adj_wgt.size(), hipMemcpyDefault, s1);
  hipMemcpyAsync(gpu_csr_in_deg, b.csr_in.deg.data(), sizeof(Weight)*b.csr_in.deg.size(), hipMemcpyDefault, s1);


  for(unsigned _ = 0; _ < numProposals; _++) {

    random_block_generator<Node, Weight> <<<num_blocks, block_size, 0, s1>>>(
      gpu_random_blocks, B
    );

    uniform_number_generator<Node, Weight> <<<num_blocks, block_size, 0, s2>>>(
      gpu_uniform_x, B
    );

    sample_neighbors<Node, Weight> <<<num_blocks, block_size, 0, s3>>>(
      gpu_sampling_neighbors1, gpu_csr_out_adj_ptr, gpu_csr_out_adj_node, gpu_csr_out_adj_wgt, 
      gpu_csr_in_adj_ptr, gpu_csr_in_adj_node, gpu_csr_in_adj_wgt, gpu_deg, B
    );

    sample_neighbors<Node, Weight> <<<num_blocks, block_size, 0, s4>>>(
      gpu_sampling_neighbors2, gpu_csr_out_adj_ptr, gpu_csr_out_adj_node, gpu_csr_out_adj_wgt,
      gpu_csr_in_adj_ptr, gpu_csr_in_adj_node, gpu_csr_in_adj_wgt, gpu_deg, B
    );

    calculate_acceptance_prob<Node, Weight> <<<num_blocks, block_size, 0, s5>>>(
      gpu_acceptance_prob, gpu_deg, B
    );

    hipDeviceSynchronize();

    propose<Node, Weight> <<<num_blocks, block_size, 0, s1>>>(
      gpu_random_blocks, gpu_sampling_neighbors1, gpu_sampling_neighbors2,
      proposed_blocks, gpu_uniform_x, gpu_acceptance_prob, gpu_deg, B
    );

    hipDeviceSynchronize();

    calculate_dS_out<Node, Weight> <<<num_blocks, block_size, 0, s1>>>(
      gpu_dS_out, gpu_csr_out_adj_ptr, gpu_csr_out_adj_node, gpu_csr_out_adj_wgt,
      gpu_csr_out_deg, gpu_csr_in_deg, B
    );

    calculate_dS_in<Node, Weight> <<<num_blocks, block_size, 0, s2>>>(
      gpu_dS_in, gpu_csr_in_adj_ptr, gpu_csr_in_adj_node, gpu_csr_in_adj_wgt,
      gpu_csr_in_deg, gpu_csr_out_deg, B
    );

    calculate_dS_new_out<Node, Weight> <<<num_blocks, block_size, 0, s3>>>(
      gpu_dS_new_out, proposed_blocks, gpu_csr_out_adj_ptr, gpu_csr_out_adj_node, 
      gpu_csr_out_adj_wgt, gpu_csr_out_deg, gpu_csr_in_deg, B
    );

    calculate_dS_new_in<Node, Weight> <<<num_blocks, block_size, 0, s4>>>(
      gpu_dS_new_in, proposed_blocks, gpu_csr_in_adj_ptr, gpu_csr_in_adj_node,
      gpu_csr_in_adj_wgt, gpu_csr_in_deg, gpu_csr_out_deg, B
    );

    hipDeviceSynchronize();

    calculate_dS_overall<Node, Weight> <<<num_blocks, block_size, 0, s4>>>(
      gpu_dS, gpu_dS_out, gpu_dS_in, gpu_dS_new_out, gpu_dS_new_in,
      gpu_csr_out_adj_ptr, gpu_csr_out_adj_node, gpu_csr_out_adj_wgt, gpu_csr_out_deg, 
      gpu_csr_in_deg, proposed_blocks, best_proposed_blocks, B
    );

    hipDeviceSynchronize();
  }

  // get the result back
  hipMemcpyAsync(&dS[0], gpu_dS, sizeof(float)*B, hipMemcpyDefault, s1);
  hipMemcpyAsync(&S[0], best_proposed_blocks, sizeof(Node)*B, hipMemcpyDefault, s1);


  hipFreeAsync(gpu_csr_out_adj_ptr, s1);
  hipFreeAsync(gpu_csr_out_adj_node, s1);
  hipFreeAsync(gpu_csr_out_adj_wgt, s1);
  hipFreeAsync(gpu_csr_out_deg, s1);
  hipFreeAsync(gpu_csr_in_adj_ptr, s1);
  hipFreeAsync(gpu_csr_in_adj_node, s1);
  hipFreeAsync(gpu_csr_in_adj_wgt, s1);
  hipFreeAsync(gpu_csr_in_deg, s1);
  hipFreeAsync(gpu_deg, s1);
  hipFreeAsync(gpu_random_blocks, s1);
  hipFreeAsync(gpu_sampling_neighbors1, s1);
  hipFreeAsync(gpu_sampling_neighbors2, s1);
  hipFreeAsync(gpu_uniform_x, s1);
  hipFreeAsync(gpu_acceptance_prob, s1);
  hipFreeAsync(proposed_blocks, s1);
  hipFreeAsync(best_proposed_blocks, s1);
  hipFreeAsync(gpu_dS_out, s1);
  hipFreeAsync(gpu_dS_in, s1);
  hipFreeAsync(gpu_dS_new_out, s1);
  hipFreeAsync(gpu_dS_new_in, s1);
  hipFreeAsync(gpu_dS, s1);
  
  hipStreamDestroy(s1);
  hipStreamDestroy(s2);
  hipStreamDestroy(s3);
  hipStreamDestroy(s4);
  hipStreamDestroy(s5);

}


template <typename Node, typename Weight>
void propose_nodal_move() {


  random_block_generator_nodal<Node, Weight> <<<num_blocks, block_size, 0, s1>>>(
    gpu_random_blocks, B, g.N
  );

  uniform_number_generator<Node, Weight> <<<num_blocks, block_size, 0, s2>>>(
    gpu_uniform_x, g.N
  );

  sample_neighbors<Node, Weight> <<<num_blocks, block_size, 0, s3>>>(
    gpu_sampling_neighbors, gpu_b_csr_out_adj_ptr, gpu_b_csr_out_adj_node, gpu_b_csr_out_adj_wgt,
    gpu_b_csr_in_adj_ptr, gpu_b_csr_in_adj_node, gpu_b_csr_in_adj_wgt, gpu_b_deg, B
  );

  sample_neighbors_nodal<Node, Weight> <<<num_blocks, block_size, 0, s4>>>(
    gpu_sampling_neighbors_nodal, gpu_g_csr_out_adj_ptr, gpu_g_csr_out_adj_node,
    gpu_g_csr_out_adj_wgt, gpu_g_csr_in_adj_ptr, gpu_g_csr_in_adj_node,
    gpu_g_csr_in_adj_wgt, gpu_g_deg, gpu_blocks, N
  );

  calculate_acceptance_prob<Node, Weight> <<<num_blocks, block_size, 0, s5>>>(
    gpu_acceptance_prob, gpu_deg, B
  );

  hipDeviceSynchronize();

  /////////////////////////////
  ////////////////////////////
  // add BS


    propose<Node, Weight> <<<num_blocks, block_size, 0, s1>>>(
      gpu_random_blocks, gpu_sampling_neighbors1, gpu_sampling_neighbors2,
      proposed_blocks, gpu_uniform_x, gpu_acceptance_prob, gpu_deg, B
    );

    hipDeviceSynchronize();

  ///////////////////////////
    ///////////////


  calculate_dS_out(float* dS_out,
                                 unsigned* csr_out_adj_ptr,
                                 Node* csr_out_adj_node,
                                 Weight* csr_out_adj_wgt,
                                 Weight* csr_out_deg,
                                 Weight* csr_in_deg,
                                 unsigned B)

  calculate_dS_in(float* dS_in,
                                unsigned* csr_in_adj_ptr,
                                Node* csr_in_adj_node,
                                Weight* csr_in_adj_wgt,
                                Weight* csr_in_deg,
                                Weight* csr_out_deg,
                                unsigned B)

}

int main (int argc, char *argv[]) {
 

  unsigned numProposals = 10;
  float blockReduction = 0.5;


  std::string FileName("../Dataset/static/lowOverlap_lowBlockSizeVar/static_lowOverlap_lowBlockSizeVar");
 
  if(argc != 2) {
    std::cerr << "usage: ./run [Number of Nodes]\n";
    std::exit(1);
  }

  int num_nodes = std::stoi(argv[1]);

  switch(num_nodes)  {
    case 1000:
      FileName += "_1000_nodes";
      break;
    case 5000:
      FileName += "_5000_nodes";
      break;
    default:
      std::cerr << "usage: ./run [Number of Nodes=1000/5000/20000/50000]\n";
      std::exit(1);
  }

  Graph g = load_graph_from_tsv<unsigned, long>(FileName);
  g.blocks.resize(g.N);
  std::iota(g.blocks.begin(), g.blocks.end(), 0);
  std::cout << "Number of nodes: " << g.N << std::endl;
  std::cout << "Number of edges: " << g.E << std::endl;

  Block<unsigned, long> b;
  unsigned B = g.N;
  unsigned bToMerges = B * blockReduction;

  initialize_block_count(g, b, B);

  std::vector<unsigned> S(B);
  std::vector<float> dS(B);
  propose_block_merge(b, B, S, dS, numProposals);


  carry_out_best_merge(g, dS, S, B, bToMerges);
  
  B -= bToMerges;



  return 0;
} 
